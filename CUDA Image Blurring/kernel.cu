
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <string>

#define CHECK(e) { int res = (e); if (res) printf("CUDA ERROR %d\n", res); }

// RGB channels
#define CHANNEL 3

// struct for image
struct Image {
	int width;
	int height;
	unsigned int bytes;
	unsigned char* img;
	unsigned char* dev_img;
};

// headers
void addBlur(Image& source, int block_size, int grid_size);
int readInpImg(const char* fname, Image& source, int& max_col_val);
int writeOutImg(const char* fname, const Image& roted, const int max_col_val);

/*
 * Kernel for generating blur from an image
 */
__global__ void rgbKernel(unsigned char* dev_source, unsigned char* dev_image, int width, int height, int grid_radius)
{
	// get block info
	int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y, bdx = blockDim.x, bdy = blockDim.y;

	// variables for accessing shared image (block co-ords)
	int row = by * bdy + ty;
	int col = bx * bdx + tx;
	// variable for accessing input image (1d image co-ords)
	int src = (row * width + col) * CHANNEL;

	extern __shared__ unsigned char shared_source[]; // create shared variable for input image

	if (col >= 0 && col < width && row >= 0 && row < height)
	{
		int index = (ty * (bdx)+tx) * CHANNEL; // get index for 1d shared memory image
		shared_source[index] = dev_source[src];
		shared_source[index + 1] = dev_source[src + 1];
		shared_source[index + 2] = dev_source[src + 2];
	}

	__syncthreads();

	int r = 0, g = 0, b = 0; // total rgb
	int count = 0; // count number of additions
	for (int i = -grid_radius; i <= grid_radius; i++) {
		for (int j = -grid_radius; j <= grid_radius; j++) {
			int filter_row = ty + j;
			int filter_col = tx + i;

			// if outside bounds of shared image, fetch from global memory
			if (filter_col >= bdx || filter_row >= bdy || filter_col < 0 || filter_row < 0)
			{
				int y = by * bdy + filter_row;
				int x = bx * bdx + filter_col;

				if (x < 0 || x >= width || y < 0 || y >= height) // make sure bounds inside image
					continue;

				int index = (y * width + x) * CHANNEL;
				r += dev_source[index];
				g += dev_source[index + 1];
				b += dev_source[index + 2];
			}
			else { // otherwise retrieve rgb from shared memory
				int index = (filter_row * bdx + filter_col) * CHANNEL;
				r += shared_source[index];
				g += shared_source[index + 1];
				b += shared_source[index + 2];
			}
			count++;
		}
	}

	// calculate and set average rgb
	dev_image[src] = r / count;
	dev_image[src + 1] = g / count;
	dev_image[src + 2] = b / count;
}

int main(int argc, char** argv)
{
	if (argc != 4)
	{
		printf("Usage: exec filename block_size grid_size\n");
		exit(1);
	}
	char* fname = argv[1];
	int blocksize = std::stoi(argv[2]); // set number of threads in block (n x n)
	int gridsize = std::stoi(argv[3]); // set total width of grid for filter

	//Read the input file
	Image source;
	int max_col_val;
	if (readInpImg(fname, source, max_col_val) != 0)  exit(1);

	// Complete the code
	addBlur(source, blocksize, gridsize);

	hipDeviceReset();

	// Write the output file
	if (writeOutImg("roted.ppm", source, max_col_val) != 0) // For demonstration, the input file is written to a new file named "roted.ppm" 
		exit(1);

	free(source.img);
    return 0;
}

void addBlur(Image &source, int block_size, int grid_size)
{
	// get/set image variables
	int width = source.width;
	int height = source.height;
	int size = width * height * 3 * sizeof(unsigned char);

	// allocate device memory
	unsigned char* dev_source;
	hipMalloc((void**)&dev_source, size);
	
	unsigned char* dev_image;
	hipMalloc((void**)&dev_image, size);

	// copy image to device
	hipMemcpy(dev_source, source.img, size, hipMemcpyHostToDevice);

	// define threads/blocks/block size
	dim3 n_threads(block_size, block_size);
	dim3 n_blocks(ceil(width/ block_size), ceil(height/ block_size));
	int shared_size = (block_size * block_size * CHANNEL);

	// record cuda event
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// execute kernel
	rgbKernel <<< n_blocks, n_threads, shared_size >>> (dev_source, dev_image, width, height, grid_size/2);

	// copy output image from device to host
	hipMemcpy(source.dev_img, dev_image, size, hipMemcpyDeviceToHost);

	// stop recording cuda event
	hipEventRecord(stop, 0);	
	hipEventSynchronize(stop);
	float t = 0;
	hipEventElapsedTime(&t, start, stop); // get elapsed time
	std::cout << "Elapsed Time: " << t << std::endl; // output time

	// clear up memory
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_source);
	hipFree(dev_image);
}

// Reads a color PPM image file (name provided), and

// saves data in the provided Image structure. 
// The max_col_val is set to the value read from the 
// input file. This is used later for writing output image. 
int readInpImg(const char* fname, Image& source, int& max_col_val) {

	FILE* src;

	if (!(src = fopen(fname, "rb")))
	{
		printf("Couldn't open file %s for reading.\n", fname);
		return 1;
	}

	char p, s;
	fscanf(src, "%c%c\n", &p, &s);
	if (p != 'P' || s != '6')   // Is it a valid format?
	{
		printf("Not a valid PPM file (%c %c)\n", p, s);
		exit(1);
	}

	fscanf(src, "%d %d\n", &source.width, &source.height);
	fscanf(src, "%d\n", &max_col_val);

	int pixels = source.width * source.height;
	source.bytes = pixels * 3;  // 3 => colored image with r, g, and b channels 
	source.img = (unsigned char*)malloc(source.bytes);
	source.dev_img = (unsigned char*)malloc(source.bytes);


	if (fread(source.img, sizeof(unsigned char), source.bytes, src) != source.bytes)
	{
		printf("Error reading file.\n");
		exit(1);
	}
	fclose(src);
	return 0;
}

// Write a color image into a file (name provided) using PPM file format.  
// Image structure represents the image in the memory. 
int writeOutImg(const char* fname, const Image& roted, const int max_col_val) {

	FILE* out;
	if (!(out = fopen(fname, "wb")))
	{
		printf("Couldn't open file for output.\n");
		return 1;
	}
	fprintf(out, "P6\n%d %d\n%d\n", roted.width, roted.height, max_col_val);
	if (fwrite(roted.dev_img, sizeof(unsigned char), roted.bytes, out) != roted.bytes)
	{
		printf("Error writing file.\n");
		return 1;
	}
	fclose(out);
	return 0;
}
